#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <windows.h>
#include <iostream>

#include "solver.h"
#include "gauss.h"

void reduce(matrix* oldMatrices, int n, matrix* newMatrices, int size);
void solve(matrix* matrices, int n, int size);
int calculate(int _size, int _log);

int main()
{
	for(int log = 1, int size = 2; size <= 64; log += 1, size *= 2)
		calculate(size, log);
}

int calculate(int _size, int _log){
	int size = _size;
	int log = _log;
	float E1 = 2;
	float E2 = 0.1;
	int matrix_no = size;
	size = (size+1)*2;
	
	matrix** matrices = (matrix**)malloc(sizeof(matrix*)*(log + 1));
	float** data = (float**)malloc(sizeof(float*)*(log + 1));

	hipMalloc((void**)&(matrices[0]), sizeof(matrix)*matrix_no);
	hipMalloc((void**)&(data[0]), matrix_no*sizeof(float)*matrix_size(size));
	hipDeviceSynchronize();
	
	init_matrices<<<matrix_no, 1>>>(matrices[0], data[0], size);
	hipDeviceSynchronize();
	
	fillLeft<<<1, size>>>(matrices[0],  E1, size);
	fillInside<<<matrix_no-1, size>>>(matrices[0]+1, E1, E2, size, matrix_no);
	hipDeviceSynchronize();
	
	int i,j;
	int n ;		
	for(i = 0,n=matrix_no; i < log; i++,n/=2){		
		hipMalloc((void**)&matrices[i+1], sizeof(matrix)*n/2);
		hipMalloc((void**)&data[i+1], sizeof(float)*matrix_size(size)*n/2);
		hipDeviceSynchronize();
		
		init_matrices<<<n/2, 1>>>(matrices[i+1], data[i + 1], size);
		hipDeviceSynchronize();
			
		reduce(matrices[i], n, matrices[i+1], size);			
	}
	
	matrix temp;
	float* temp_data = (float*)malloc(matrix_size(size) * sizeof(float));
	hipMemcpy(&temp, matrices[log], sizeof(matrix), hipMemcpyDeviceToHost);
	hipDeviceSynchronize();
	hipMemcpy(temp_data, temp.ur, sizeof(float) * matrix_size(size), hipMemcpyDeviceToHost);
	float* target = temp.ur;
	hipDeviceSynchronize();
	
	temp.ur = temp_data;
	temp.ul = temp.ur + size*size;
	temp.lr = temp.ul + size*size;
	temp.ll = temp.lr + size*size;
	temp.ub = temp.ll + size*size;
	temp.lb = temp.ub + size;
	
	gauss(&temp, size);
	double sum_of_stuff = 0;
	for(int i = 1; i < size; i+=2)
		sum_of_stuff += temp.lb[i];
	std::cout << sum_of_stuff/size << " ";
	
	hipMemcpy(target, temp_data, sizeof(float) * matrix_size(size), hipMemcpyHostToDevice);
	hipDeviceSynchronize();
	
	matrix top_left, top_right;
	hipMemcpy(&top_left, matrices[log-1], sizeof(matrix), hipMemcpyDeviceToHost);
	hipMemcpy(&top_right, matrices[log-1]+1, sizeof(matrix), hipMemcpyDeviceToHost);
	hipDeviceSynchronize();
	hipMemcpy(top_left.ul, temp.ul, sizeof(float) * size * size, hipMemcpyHostToDevice);
	hipMemcpy(top_left.ur, temp.ur, sizeof(float) * size * size, hipMemcpyHostToDevice);	
	hipMemcpy(top_left.ub, temp.ub, sizeof(float) * size, hipMemcpyHostToDevice);
	hipMemcpy(top_right.ll, temp.ll, sizeof(float) * size * size, hipMemcpyHostToDevice);
	hipMemcpy(top_right.lr, temp.lr, sizeof(float) * size * size, hipMemcpyHostToDevice);	
	hipMemcpy(top_right.lb, temp.lb, sizeof(float) * size, hipMemcpyHostToDevice);
	hipDeviceSynchronize();
	
	free(temp_data);
	
		
	for(i = log-1,n=2; i >= 0; i--,n*=2){
		solve(matrices[i], n, size);

		if(i > 0)
			for(j = 0; j < n; j+=2){
				copyBUpper<<<1, size>>>(matrices[i]+j, matrices[i-1]+j*2);
				copyBLower<<<1, size>>>(matrices[i]+j, matrices[i]+j+1, matrices[i-1]+j*2+1, size);
				copyBUpper<<<1, size>>>(matrices[i]+j, matrices[i]+j+1, matrices[i-1]+j*2+2, size);
				copyBLower<<<1, size>>>(matrices[i]+j+1, matrices[i-1]+j*2+3);								
			}
				
		hipDeviceSynchronize();
	}
	
	float* values;
	hipMalloc((void**)&values, sizeof(float)*(matrix_no+1)*size);
	hipDeviceSynchronize();
	extractResults<<<1, matrix_no/2>>>(matrices[0], values, size);
	hipDeviceSynchronize();
	float* disps = (float*)malloc((matrix_no+1)*size*sizeof(float));
	hipMemcpy(disps, values, (matrix_no+1)*size*sizeof(float), hipMemcpyDeviceToHost);	
	hipDeviceSynchronize();
	
	float* contributions;
	hipMalloc((void**)&contributions, sizeof(float)*matrix_no*size);
	calculateEnergy<<<matrix_no, size>>>(E1, E2, values, contributions, size);
	hipDeviceSynchronize();
	
	float* results = (float*)malloc(matrix_no*size*sizeof(float));
	hipMemcpy(results, contributions, matrix_no*size*sizeof(float), hipMemcpyDeviceToHost);
	hipDeviceSynchronize();
	
	double energy = 0;
	double mean = 0;
	for(int i = 0; i < matrix_no*size; i+= 1)
	{
		mean += disps[i];
		energy += results[i];
	}
	std::cout << energy <<  " " << mean /(matrix_no*size) << "\n";
	
	for(int i =0; i < log+1; i++)
	{
		hipFree(data[i]);
		hipFree(matrices[i]);
	}
	hipDeviceSynchronize();
	
	return 0;
}

void reduce(matrix* oldMatrices, int n, matrix* newMatrices, int size){
	int i;
	
	for(i = 0; i < n; i += 2){
		copyUpperLeft<<<size,size>>>(oldMatrices+i, newMatrices+i/2);
		copyLowerRight<<<size,size>>>(oldMatrices+i+1, newMatrices+i/2);
		copyBUpper<<<1,size>>>(oldMatrices+i, newMatrices+i/2);
		copyBLower<<<1,size>>>(oldMatrices+i+1, newMatrices+i/2);
	}
	
	float* coeffs;
	hipMalloc((void**)&coeffs, sizeof(float)*3*size*n);
	hipDeviceSynchronize();
	init_vector<<<3*n, 1>>>(coeffs, size);
	hipDeviceSynchronize();
		
	int row;
	for(row = 0; row < size; row++){
		float* dElement;
		hipMalloc((void**)&dElement, sizeof(float)*n/2);
		hipDeviceSynchronize();
		calculateElement<<<1,n/2>>>(dElement, row, oldMatrices, size);
		hipDeviceSynchronize();
		
		//Calculate coefficients
		for(i = 0; i < n; i += 2){
			countCoeffsUpper<<<1, size>>>(dElement+i/2, oldMatrices+i, row, coeffs+(3*size)*i, size);
			countCoeffsCenter<<<1,size-row-1>>>(dElement+i/2, oldMatrices+i, oldMatrices+i+1, row, coeffs+(3*size)*i+size, size);
			countCoeffsLower<<<1, size>>>(dElement+i/2, oldMatrices+i+1, row, coeffs+(3*size)*i+2*size, size);
		}
		hipDeviceSynchronize();
		
		for(i = 0; i < n; i += 2){
			updateUpperLeft<<<size, size>>>(row, oldMatrices+i, newMatrices+i/2, coeffs+(3*size)*i,  size);
			updateUpperCenter<<<size, size>>>(row, oldMatrices+i, oldMatrices+i+1, coeffs+(3*size)*i,  size);
			updateUpperRight<<<size, size>>>(row, oldMatrices+i+1, newMatrices+i/2, coeffs+(3*size)*i,  size);
			updateBUpper<<<1, size>>>(row, oldMatrices+i, oldMatrices+i+1, newMatrices+i/2, coeffs+(3*size)*i,  size);
			
			updateLeftCenter<<<size-row-1, size>>>(row, oldMatrices+i, coeffs+(3*size)*i+size,  size);
			updateCenter<<<size-row-1, size>>>(row, oldMatrices+i, oldMatrices+i+1, coeffs+(3*size)*i+size,  size);
			updateRightCenter<<<size-row-1, size>>>(row, oldMatrices+i+1, coeffs+(3*size)*i+size,  size);
			updateBCenter<<<1, size-row-1>>>(row, oldMatrices+i, oldMatrices+i+1, coeffs+(3*size)*i+size,  size);
			
			updateLowerLeft<<<size, size>>>(row, oldMatrices+i, newMatrices+i/2, coeffs+(3*size)*i+2*size,  size);
			updateLowerCenter<<<size, size>>>(row, oldMatrices+i, oldMatrices+i+1, coeffs+(3*size)*i+2*size,  size);
			updateLowerRight<<<size, size>>>(row, oldMatrices+i+1, newMatrices+i/2, coeffs+(3*size)*i+2*size,  size);
			updateBLower<<<1, size>>>(row, oldMatrices+i, oldMatrices+i+1, newMatrices+i/2, coeffs+(3*size)*i+2*size,  size);
			
		}
		
		hipDeviceSynchronize();
	}
	
	hipFree(coeffs);
}

void solve(matrix* matrices, int n, int size){
	int i, j;
	
	for(i = 0; i < size; i++){
		for(j = 0; j < n; j+=2)
			backwardsSubstitutionRight<<<1, size>>>(i, matrices+j, matrices+j+1, size);
		hipDeviceSynchronize();
		for(j = 0; j < n; j+=2)
			backwardsSubstitutionLeft<<<1,size>>>(i, matrices+j, matrices+j+1, size);
		hipDeviceSynchronize();
	}
	
	for(i = 0; i < size; i++){
		for(j = 0; j < n; j+=2)
			backwardsSubstitutionCenter<<<1, size-i-1>>>(size-i-1, matrices+j, matrices+j+1, size);
		hipDeviceSynchronize();
	}
	
}

void fix_top(matrix* matrix, int size)
{
}