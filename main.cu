#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <windows.h>
#include <iostream>

#include "solver.h"

void reduce(matrix* oldMatrices, int n, matrix* newMatrices, int size);
void solve(matrix* matrices, int n, int size);

int main(){
	int size = 4;
	int log = 2;
	float E1 = 1;
	float E2 = 1;
	int matrix_no = size;
	size = (size+1)*2;
	
	matrix** matrices = (matrix**)malloc(sizeof(matrix*)*(log + 1));
	float** data = (float**)malloc(sizeof(float*)*(log + 1));

	hipMalloc((void**)&(matrices[0]), sizeof(matrix)*matrix_no);
	hipMalloc((void**)&(data[0]), matrix_no*sizeof(float)*matrix_size(size));
	hipDeviceSynchronize();
	
	init_matrices<<<matrix_no, 1>>>(matrices[0], data[0], size);
	hipDeviceSynchronize();
	
	fillLeft<<<1, size>>>(matrices[0],  E1, size);
	fillInside<<<matrix_no-1, size>>>(matrices[0]+1, E1, E2, size, matrix_no);
	hipDeviceSynchronize();

		printDeviceMatrix(matrices[0], size);
		
	int i,j;
	int n ;		
	for(i = 0,n=matrix_no; i < log; i++,n/=2){		
		hipMalloc((void**)&matrices[i+1], sizeof(matrix)*n/2);
		hipMalloc((void**)&data[i+1], sizeof(float)*matrix_size(size)*n/2);
		hipDeviceSynchronize();
		
		init_matrices<<<n/2, 1>>>(matrices[i+1], data[i + 1], size);
		hipDeviceSynchronize();
			
		reduce(matrices[i], n, matrices[i+1], size);
	}
		
	for(i = log-1,n=2; i >= 0; i--,n*=2){
		solve(matrices[i], n, size);
		if(i > 0)
			for(j = 0; j < n; j+=2){
				copyBUpper<<<1, size>>>(matrices[i]+j, matrices[i-1]+j*2);
				copyBLower<<<1, size>>>(matrices[i]+j, matrices[i]+j+1, matrices[i-1]+j*2+1, size);
				copyBUpper<<<1, size>>>(matrices[i]+j, matrices[i]+j+1, matrices[i-1]+j*2+2, size);
				copyBLower<<<1, size>>>(matrices[i]+j+1, matrices[i-1]+j*2+3);								
			}
		
		hipDeviceSynchronize();		
	}
	
	float* results;
	hipMalloc((void**)&results, sizeof(float)*matrix_no*size);
	hipDeviceSynchronize();
	extractResults<<<1, matrix_no/2>>>(matrices[0], results, size);
	hipDeviceSynchronize();	
	
	printDeviceVector(results, (matrix_no+1)*size);
	
	for(int i =0; i < log+1; i++)
	{
		hipFree(data[i]);
		hipFree(matrices[i]);
	}
	hipDeviceSynchronize();
	
	return 0;
}

void reduce(matrix* oldMatrices, int n, matrix* newMatrices, int size){
	int i;
	
	for(i = 0; i < n; i += 2){
		copyUpperLeft<<<size,size>>>(oldMatrices+i, newMatrices+i/2);
		copyLowerRight<<<size,size>>>(oldMatrices+i+1, newMatrices+i/2);
		copyBUpper<<<1,size>>>(oldMatrices+i, newMatrices+i/2);
		copyBLower<<<1,size>>>(oldMatrices+i+1, newMatrices+i/2);
	}
	
	float* coeffs;
	hipMalloc((void**)&coeffs, sizeof(float)*3*size*n);
	hipDeviceSynchronize();
	init_vector<<<3*n, 1>>>(coeffs, size);
	hipDeviceSynchronize();
		
	int row;
	for(row = 0; row < size; row++){
		float* dElement;
		hipMalloc((void**)&dElement, sizeof(float)*n/2);
		hipDeviceSynchronize();
		calculateElement<<<1,n/2>>>(dElement, row, oldMatrices, size);
		hipDeviceSynchronize();
		
		//Calculate coefficients
		for(i = 0; i < n; i += 2){
			countCoeffsUpper<<<1, size>>>(dElement+i/2, oldMatrices+i, row, coeffs+(3*size)*i, size);
			countCoeffsCenter<<<1,size-row-1>>>(dElement+i/2, oldMatrices+i, oldMatrices+i+1, row, coeffs+(3*size)*i+size, size);
			countCoeffsLower<<<1, size>>>(dElement+i/2, oldMatrices+i+1, row, coeffs+(3*size)*i+2*size, size);
		}
		hipDeviceSynchronize();
		
		for(i = 0; i < n; i += 2){
			updateUpperLeft<<<size, size>>>(row, oldMatrices+i, newMatrices+i/2, coeffs+(3*size)*i,  size);
			updateUpperCenter<<<size, size>>>(row, oldMatrices+i, oldMatrices+i+1, coeffs+(3*size)*i,  size);
			updateUpperRight<<<size, size>>>(row, oldMatrices+i+1, newMatrices+i/2, coeffs+(3*size)*i,  size);
			updateBUpper<<<1, size>>>(row, oldMatrices+i, oldMatrices+i+1, newMatrices+i/2, coeffs+(3*size)*i,  size);
			
			updateLeftCenter<<<size-row-1, size>>>(row, oldMatrices+i, coeffs+(3*size)*i+size,  size);
			updateCenter<<<size-row-1, size>>>(row, oldMatrices+i, oldMatrices+i+1, coeffs+(3*size)*i+size,  size);
			updateRightCenter<<<size-row-1, size>>>(row, oldMatrices+i+1, coeffs+(3*size)*i+size,  size);
			updateBCenter<<<1, size-row-1>>>(row, oldMatrices+i, oldMatrices+i+1, coeffs+(3*size)*i+size,  size);
			
			updateLowerLeft<<<size, size>>>(row, oldMatrices+i, newMatrices+i/2, coeffs+(3*size)*i+2*size,  size);
			updateLowerCenter<<<size, size>>>(row, oldMatrices+i, oldMatrices+i+1, coeffs+(3*size)*i+2*size,  size);
			updateLowerRight<<<size, size>>>(row, oldMatrices+i+1, newMatrices+i/2, coeffs+(3*size)*i+2*size,  size);
			updateBLower<<<1, size>>>(row, oldMatrices+i, oldMatrices+i+1, newMatrices+i/2, coeffs+(3*size)*i+2*size,  size);
			
		}
		
		hipDeviceSynchronize();
	}
	
	hipFree(coeffs);
}

void solve(matrix* matrices, int n, int size){
	int i, j;
	
	for(i = 0; i < size; i++){
		for(j = 0; j < n; j+=2)
			backwardsSubstitutionRight<<<1, size>>>(i, matrices+j, matrices+j+1, size);
		hipDeviceSynchronize();
		for(j = 0; j < n; j+=2)
			backwardsSubstitutionLeft<<<1,size>>>(i, matrices+j, matrices+j+1, size);
		hipDeviceSynchronize();
	}
	
	for(i = 0; i < size; i++){
		for(j = 0; j < n; j+=2)
			backwardsSubstitutionCenter<<<1, size-i-1>>>(size-i-1, matrices+j, matrices+j+1, size);
		hipDeviceSynchronize();
	}
	
}